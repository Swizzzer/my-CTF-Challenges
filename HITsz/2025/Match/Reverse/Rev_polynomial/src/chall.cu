#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define N (1 << 9)
#define P 998244353

typedef struct {
    int x;
} mint;

__constant__ mint c_w[N];
__constant__ mint c_inv[N];
__constant__ int c_secret1[18];

__host__ __device__ mint mint_create(int x) {
    mint m;
    m.x = x % P;
    if (m.x < 0) m.x += P;
    return m;
}

__host__ __device__ mint mint_add(mint a, mint b) {
    int res = a.x + b.x;
    return mint_create(res);
}

__host__ __device__ mint mint_mul(mint a, mint b) {
    long long product = (long long)a.x * b.x;
    return mint_create((int)(product % P));
}

__host__ __device__ mint mint_pow(mint a, int k) {
    if (k == 0) return mint_create(1);
    mint result = a;
    for (int i = 1; i < k; ++i) {
        result = mint_mul(result, a);
    }
    return result;
}

void init_host(mint* w, mint* inv) {
    // w初始化
    w[N/2] = mint_create(1);
    mint g_base = mint_create(3);
    int exponent = P / N;
    mint g = mint_create(1);
    for (int i = 0; i < exponent; ++i) {
        g = mint_mul(g, g_base);
    }

    for (int i = N/2 + 1; i < N; i++) {
        w[i] = mint_mul(w[i-1], g);
    }
    for (int i = N/2 - 1; i > 0; --i) {
        w[i] = w[i << 1];
    }

    // inv初始化
    inv[0] = mint_create(1);
    inv[1] = mint_create(1);
    for (int i = 2; i < N; i++) {
        mint temp = mint_create(P - P/i);
        inv[i] = mint_mul(inv[P % i], temp);
    }
}

__global__ void verify_flag_kernel(mint* d_a, int len, int* d_results) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (x > 18) return;

    mint ans = mint_create(0);
    mint tmp = mint_create(1);
    mint x_mint = mint_create(x);

    for (int i = 0; i < len; i++) {
        mint term = mint_mul(d_a[i], tmp);
        ans = mint_add(ans, term);
        tmp = mint_mul(tmp, x_mint);
    }

    d_results[x-1] = (ans.x == c_secret1[x-1]) ? 1 : 0;
}

int challenge() {
    char input[19];
    printf("Please input the flag: ");
    if (scanf("%18s", input) != 1) {
        printf("Invalid input!\n");
        return 1;
    }
    
    int len = strlen(input);
    if (len != 18) {
        printf("Failed: Incorrect length!\n");
        return 1;
    }

    // 主机数据
    mint* h_a = (mint*)malloc(len * sizeof(mint));
    for (int i = 0; i < len; i++) {
        h_a[i] = mint_create((int)input[i]);
    }

    mint* d_a;
    int* d_results;
    int* h_results = (int*)malloc(18 * sizeof(int));
    hipMalloc(&d_a, len * sizeof(mint));
    hipMalloc(&d_results, 18 * sizeof(int));

    // 数据拷贝到设备
    hipMemcpy(d_a, h_a, len * sizeof(mint), hipMemcpyHostToDevice);

    // 启动核函数
    dim3 block(32);
    dim3 grid((18 + block.x - 1) / block.x);
    verify_flag_kernel<<<grid, block>>>(d_a, len, d_results);

    // 回传
    hipMemcpy(h_results, d_results, 18 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 18; i++) {
        if (!h_results[i]) {
            printf("Failed at position %d!\n", i+1);
            free(h_a);
            free(h_results);
            hipFree(d_a);
            hipFree(d_results);
            return 1;
        }
    }

    printf("Congratulations! You got the flag! Remember to wrap it with HITCTF{}\n");

    // 清理资源
    free(h_a);
    free(h_results);
    hipFree(d_a);
    hipFree(d_results);
    return 0;
}

int main() {
    mint h_w[N], h_inv[N];
    init_host(h_w, h_inv);
    hipMemcpyToSymbol(HIP_SYMBOL(c_w), h_w, N * sizeof(mint));
    hipMemcpyToSymbol(HIP_SYMBOL(c_inv), h_inv, N * sizeof(mint));

    const int secret1[18] = {1828,30029024,675933036,307266195,441725700,908486918,872572725,462684583,631964733,930026310,883382448,939457745,913070734,152279109,617088314,617509926,518595840,652243173};
    hipMemcpyToSymbol(HIP_SYMBOL(c_secret1), secret1, 18 * sizeof(int));

    printf("Can you recover the secret of polynomial?\n");
    return challenge();
}
